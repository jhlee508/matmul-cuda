#include <cstdio>
#include <hipblas.h>

#include "matmul.cuh"
#include "kernels.cuh"

#include "common.h"


static float *A_gpu, *B_gpu, *C_gpu;
static hipblasHandle_t handle;

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  /* Naive */
  // dim3 gridDim(CEIL_DIV(N, 32), CEIL_DIV(M, 32));
  // dim3 blockDim(32, 32);
  // naive_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  /* Global Memory Coalescing */
  // dim3 gridDim(CEIL_DIV(N, 32), CEIL_DIV(M, 32));
  // dim3 blockDim(32, 32);
  // gmem_coalescing_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  /* Shared Memory Caching */
  // dim3 gridDim(CEIL_DIV(N, SMEM_BS), CEIL_DIV(M, SMEM_BS));
  // dim3 blockDim(SMEM_BS, SMEM_BS);
  // smem_caching_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  /* Blocktiling 1D */
  // dim3 gridDim(CEIL_DIV(N, BLOCKTILING_1D_BN), CEIL_DIV(M, BLOCKTILING_1D_BM));
  // dim3 blockDim(BLOCKTILING_1D_BN, BLOCKTILING_1D_BM / BLOCKTILING_1D_TM);
  // blocktiling_1d_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  /* Blocktiling 1D (GMEM Coalescing) */
  // dim3 gridDim(CEIL_DIV(N, BLOCKTILING_1D_BN), CEIL_DIV(M, BLOCKTILING_1D_BM));
  // dim3 blockDim(BLOCKTILING_1D_BN, BLOCKTILING_1D_BM / BLOCKTILING_1D_TM);
  // blocktiling_1d_kernel_v2<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  /* Blocktiling 2D */
  // dim3 gridDim(CEIL_DIV(N, BLOCKTILING_2D_BN), CEIL_DIV(M, BLOCKTILING_2D_BM));
  // dim3 blockDim(BLOCKTILING_2D_BN / BLOCKTILING_2D_TN, BLOCKTILING_2D_BM / BLOCKTILING_2D_TM);
  // blocktiling_2d_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  /* Blocktiling 2D (Vectorized) */
  dim3 gridDim(CEIL_DIV(N, BLOCKTILING_2D_BN), CEIL_DIV(M, BLOCKTILING_2D_BM));
  dim3 blockDim(BLOCKTILING_2D_BN / BLOCKTILING_2D_TN, BLOCKTILING_2D_BM / BLOCKTILING_2D_TM);
  blocktiling_2d_vec_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cublas(float *_A, float *_B, float *_C, int M, int N, int K) {
  const float one = 1, zero = 0;
  CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &one,
                           B_gpu, N, A_gpu, K, &zero, C_gpu, N));
  // CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &one,
  //                           B_gpu, HIP_R_32F, N, A_gpu, HIP_R_32F, K, &zero, C_gpu, HIP_R_32F, N,
  //                           HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_initialize(float *_A, float *_B, int M, int N, int K) { 
  CHECK_CUDA(hipMalloc((void **) &A_gpu, sizeof(float) * M * K));
  CHECK_CUDA(hipMalloc((void **) &B_gpu, sizeof(float) * K * N));
  CHECK_CUDA(hipMalloc((void **) &C_gpu, sizeof(float) * M * N));

  CHECK_CUDA(
    hipMemcpy(A_gpu, _A, sizeof(float) * M * K, hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(B_gpu, _B, sizeof(float) * K * N, hipMemcpyHostToDevice));
}

void cublas_initialize() {
  CHECK_CUBLAS(hipblasCreate(&handle));
  // CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH)); /* To enable TC */
}

void matmul_finalize(float *_C, int M, int N, int K) {
  CHECK_CUDA(
    hipMemcpy(_C, C_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));
}

void cublas_finalize() {
  CHECK_CUBLAS(hipblasDestroy(handle));
}
