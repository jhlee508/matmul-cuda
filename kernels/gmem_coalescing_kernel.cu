
#include <hip/hip_runtime.h>
__global__ void gmem_coalescing_kernel(float *A, float *B, float *C, int M, int N, int K) {
	const int Col = blockIdx.x * blockDim.x + threadIdx.x;
	const int Row = blockIdx.y * blockDim.y + threadIdx.y;

	float tmp = 0.f;

	for (int k = 0; k < K; k++) {
		tmp += A[Row * K + k] * B[k * N + Col];
	}
	C[Row * N + Col] = tmp;
}
  