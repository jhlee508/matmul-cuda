
#include <hip/hip_runtime.h>
__global__ void naive_kernel(float *A, float *B, float *C, int M, int N, int K) {
	const int Row = blockIdx.x * blockDim.x + threadIdx.x;
	const int Col = blockIdx.y * blockDim.y + threadIdx.y;

	float tmp = 0.f;

	for (int k = 0; k < K; k++) {
		tmp += A[Row * K + k] * B[k * N + Col];
	}
	C[Row * N + Col] = tmp;
}